/*
This version of my sudoku solver will make use of cuda to attemp to gain speedups
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
//#include <chrono>
#define boardSize 81
#define sideSize 9

using namespace std;

struct Board{
	int values[81];
	bool isFixedValue[81];
	bool isPossibleValue[81][9];
	//int coordinates;
};

void getFileName(char fileName[]){
	cout << "Please give the file location of your sudoku board."<<endl;
	cin.getline(fileName, 255);
	return;
}

bool getBoard(char fileName[], Board &mainBoard){
	ifstream file(fileName);//input can only occur on host this can't be parallelized
	if(file.fail())
		return false;
	char temp = '0';
	for (int i = 0; i < boardSize; i++){
		file >> temp;
		mainBoard.values[i] = (int)((int)temp - (int)'0');
		//cout << mainBoard.values[i] << ",";
		if(mainBoard.values[i] > 0){
			mainBoard.isFixedValue[i] = true;
		}
		else{
			mainBoard.isFixedValue[i] = false;
			mainBoard.values[i] = (mainBoard.values[i] / -1);
		}
	}
	file.close();
	return true;
}	

__global__ void rowCheckKernel(const int board[], const int row, const int value, bool *hasDuplicates)
{
	int i = (row*sideSize) + threadIdx.x;
	if(value == board[i])
		*hasDuplicates = true;//for some reason this doesn't want to pass back the correct value, always the same one
}

bool anyDuplicates(int board[], int coordinates, int value)
{
	int row = coordinates / sideSize;
	int column = coordinates % sideSize;
	//dim3 grid(sideSize);
	//bool *hasDuplicates = false;
	//rowCheckKernel<<<1,grid>>>(board,row,value,hasDuplicates);
	//bool test = &hasDuplicates;
	/*if(test){
		return test;
	}*/
	for(int i = row * sideSize; i < ((row*sideSize)+sideSize); i++)
	{
		if(value == board[i]){
			return true;
		}
	}
	for(int i = column; i < boardSize; i+=sideSize)
	{
		if(value == board[i])
			return true;
	}
	while(column%3 != 0)
		column--;
	while(row%3 != 0)
		row--;
	for(int r = row; r < (row+3); r++)
	{
		for(int c = column; c < (column+3); c++)
		{
			if(value == board[((r*9)+c)])
				return true;
		}
	}
	return false;
}

__global__ void validateBoardKernel(Board board, bool &isValid)
{
	int tempValue = board.values[threadIdx.x];
	board.values[threadIdx.x] = 0;
	if(tempValue != 0 /*&& anyDuplicates(board.values, threadIdx.x, tempValue)*/){
		isValid = false;
	}
}

bool validateBoard(Board board){//when I attempted to paralellize this it would
	for(int i = 0; i < boardSize; i++){
		int tempValue = board.values[i];
		board.values[i] = 0;
		if(tempValue != 0 && anyDuplicates(board.values, i, tempValue)){
			cout<<"ERROR: Duplicate value '"
			    << tempValue;
			return false;
		}
		board.values[i] = tempValue;
	}
	return true;
}

void displayBoard(Board board)
{//this cannot be implemented in parallel I don't think, everything I found said input output had to occur on host
	ofstream results("resultsParallel.txt");
	for(int i = 0; i < boardSize; i++){
		results<<board.values[i] <<",";
		if(i%sideSize == 8)
			results<<endl;
	}
	results.close();
}

__global__ void isSolvedKernel(const Board board, bool &solved)
{
	if(board.values[threadIdx.x] == 0)
		solved = true;
}

bool isSolved(Board board)
{
	bool solved = true;
	dim3 grid(boardSize);
	isSolvedKernel<<<1, grid>>>(board,solved);
	return solved;
}

bool canChange(Board mainBoard, int coordinates, int value)
{//this function has no behavior worth parallelizing
	if(value==0)
		return true;
	if(mainBoard.isFixedValue[coordinates])
		return false;
	if(anyDuplicates(mainBoard.values, coordinates, value))
		return false;
	return true;
}

__global__ void checkPossiblesKernel(Board &newBoard, bool &noChanges)
{
	int possibles = 0;
	int value;
	if(!newBoard.isFixedValue[threadIdx.x]){
		for(int guess = 1; guess <= sideSize; guess++){
			if(/*canChange(newBoard, threadIdx.x, guess)can't use host function from global*/guess==0 || !newBoard.isFixedValue[threadIdx.x]){
				value = guess;
				newBoard.isPossibleValue[threadIdx.x][guess-1] = true;
				possibles++;
			}
			else
				newBoard.isPossibleValue[threadIdx.x][guess-1] = false;
		}
	}
	if(possibles==1){
		newBoard.values[threadIdx.x] = value;
		newBoard.isFixedValue[threadIdx.x] = true;
		noChanges = false;
	}
}

bool checkPossibles(Board &newBoard)
{
	bool noChanges;
	do{
		noChanges = true;
		for(int i = 0; i < boardSize; i++){
			int possibles = 0;
			int value;
			if(!newBoard.isFixedValue[i]){
				for(int guess = 1; guess <= 9; guess++){
					if(canChange(newBoard, i, guess)){
						value = guess;
						newBoard.isPossibleValue[i][guess-1] = true;
						possibles++;
					}
					else
						newBoard.isPossibleValue[i][guess-1] = false;
				}
			}
			if(possibles==1){
				newBoard.values[i] = value;
				newBoard.isFixedValue[i] = true;
				noChanges = false;
			}
		}
	}while(noChanges == false);
	if(isSolved(newBoard))
		return true;
	return false;		
}
/*dfs is very difficult to do in parallel
I couldn't figure out how to do it
__global__ void dfs(Board &board)
{
	
}

Board parallelBrute(Board board)
{
	
}
*/
Board recursiveBrute(Board board, int startPosition)
{
	while(startPosition < boardSize && board.isFixedValue[startPosition] && board.values[startPosition] != 0)
		startPosition++;
	if(startPosition >= boardSize)
		return board;
	for(int guess = 1; guess <= 9; guess++){
		Board tempBoard = board;
		if(board.isPossibleValue[startPosition][guess-1] && canChange(board,startPosition,guess)){
			board.values[startPosition] = guess;
			tempBoard = recursiveBrute(board, startPosition+1);
		}
		if(isSolved(tempBoard))
			return tempBoard;
	}
	board.values[startPosition] = 0;
	return board;
}

__global__ void solveKernel(Board &newBoard, bool &noChanges)
{
	for(int guess = 1; guess <= sideSize; guess++){
		int total = 0;
		for(int iRow = threadIdx.x; iRow < (threadIdx.x+sideSize); iRow++){
			if(!newBoard.isFixedValue[iRow])
				total += newBoard.isPossibleValue[iRow][guess-1];
		}
		if(total == 1){
			for(int iRow = threadIdx.x; iRow <(threadIdx.x + sideSize); iRow++){
				if(newBoard.isPossibleValue[iRow][guess-1] && !newBoard.isFixedValue[iRow]){
					newBoard.values[iRow] = guess;
					newBoard.isFixedValue[iRow] = true;
					noChanges = false;
				}
			}		
		}
	}
}

void solve(Board &newBoard)
{
	bool noChanges;
	do
	{
		noChanges = true;
		if(checkPossibles(newBoard))
			return;
		for(int i = 0; i < boardSize; i += sideSize){
			for(int guess = 1; guess <= 9; guess++){
				int total = 0;
				for(int iRow = i; iRow < (i+9); iRow++){
					if(!newBoard.isFixedValue[iRow])
						total += newBoard.isPossibleValue[iRow][guess-1];
				}
				if(total == 1){
					for(int iRow = i; iRow <(i+9); iRow++){
						if(newBoard.isPossibleValue[iRow][guess-1] && !newBoard.isFixedValue[iRow]){
							newBoard.values[iRow] = guess;
							newBoard.isFixedValue[iRow] = true;
							noChanges = false;
						}
					}
				}
			}
		}
		if(checkPossibles(newBoard))
			return;

		for(int i = 0; i < sideSize; i++){
			for(int guess = 1; guess <= 9; guess++){
				int total = 0;
				for(int iColumn = i; iColumn < boardSize; iColumn += sideSize){
					if(!newBoard.isFixedValue[iColumn]){
						total += newBoard.isPossibleValue[iColumn][guess-1];
					}
				}
				if(total == 1){
					for(int iColumn = i; iColumn < boardSize; iColumn += sideSize){
						if(newBoard.isPossibleValue[iColumn][guess-1] && !newBoard.isFixedValue[iColumn]){
							newBoard.values[iColumn] = guess;
							newBoard.isFixedValue[iColumn] = true;
							noChanges = false;
						}
					}
				}
			}
		}
		
		if(checkPossibles(newBoard))
			return;
		for(int r = 0; r < boardSize; r+= sideSize*3){
			for(int c = 0; c < sideSize; c += 3){
				int i = r+c;
				for(int guess = 1; guess <= 9; guess++){
					int total = 0;
					for(int iBlockR = i; iBlockR < (i+sideSize*3); iBlockR += sideSize){
						for(int iBlockC = iBlockR; iBlockC < (iBlockR + 3); iBlockC++){
							if(!newBoard.isFixedValue[iBlockC])
								total += newBoard.isPossibleValue[iBlockC][guess-1];
						}
					}
					if(total == 1){
						for(int iBlockR = i; iBlockR < (i+sideSize*3); iBlockR += sideSize){
							for(int iBlockC = iBlockR; iBlockC < (iBlockR + 3); iBlockC++){
								if(newBoard.isPossibleValue[iBlockC][guess-1] && !newBoard.isFixedValue[iBlockC]){
									newBoard.values[iBlockC] = guess;
									newBoard.isFixedValue[iBlockC] = true;
									noChanges = false;
								}
							}
						}
					}
				}
			}
		}
		if(checkPossibles(newBoard))
			return;
	}while(noChanges == false);
	if(!isSolved(newBoard))
		newBoard = recursiveBrute(newBoard, 0);
}

int main(int argc, char *argv[]){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	char fileName[256];
	Board mainBoard;
	getFileName(fileName);
	if(!getBoard(fileName,mainBoard))
		cout <<"Error with filename";
	else if(!validateBoard(mainBoard)){
		displayBoard(mainBoard);
		return 0;
	}
	solve(mainBoard);
	displayBoard(mainBoard);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout <<"Solve time took " << milliseconds << "ms"<<endl;
	return 0;
}















